// src/vector_add.cu

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Template CUDA Kernel for vector addition
template <typename T>
__global__ void vectorAddKernel(const T* A, const T* B, T* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

// Host function template to perform vector addition
template <typename T>
int vector_add(const T* A, const T* B, T* C, int numElements) {
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(T);

    // Allocate device memory
    T* d_A = nullptr;
    err = hipMalloc((void**)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    T* d_B = nullptr;
    err = hipMalloc((void**)&d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_A);
        return -1;
    }

    T* d_C = nullptr;
    err = hipMalloc((void**)&d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        return -1;
    }

    // Copy input data from host to device
    err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return -1;
    }

    err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return -1;
    }

    // Launch the CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAddKernel<T><<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAddKernel (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return -1;
    }

    // Wait for GPU to finish before accessing on host
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %s after launching kernel!\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return -1;
    }

    // Copy the result from device to host
    err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return -1;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0; // Success
}

// Explicit instantiation of the template for float and double
extern "C" {

    // float version
    int vector_add_f32(const float* A, const float* B, float* C, int numElements) {
        return vector_add<float>(A, B, C, numElements);
    }

    // double version
    int vector_add_f64(const double* A, const double* B, double* C, int numElements) {
        return vector_add<double>(A, B, C, numElements);
    }

} // extern "C"
